#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/

#include <iostream>
#include "utils.h"

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals,
	       int numBins)
{
        extern __shared__ unsigned int temp[];
	temp[threadIdx.x] = 0;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx >= numVals) { return; }
        atomicAdd(&temp[vals[idx]], 1);
        __syncthreads();
        
        for(int pos = threadIdx.x; pos < numBins; pos+=blockDim.x)
        {
                atomicAdd(&histo[pos], temp[pos]);
        }
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo, //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  const int THREADS = 1024;
  const int BLOCKS = numElems / THREADS;
  yourHisto<<<BLOCKS, THREADS, numBins * sizeof(unsigned int)>>>(d_vals, d_histo, numElems, numBins);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
